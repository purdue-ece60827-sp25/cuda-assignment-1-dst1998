#include "hip/hip_runtime.h"

#include "cudaLib.cuh"

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ 
void saxpy_gpu (float* x, float* y, float scale, int size) {
	//	Insert GPU SAXPY kernel code here
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= size) return;
    y[i] = scale * x[i] + y[i];
}

int runGpuSaxpy(int vectorSize) {

	std::cout << "Hello GPU Saxpy!\n";

	//	Insert code here
	// std::cout << "Lazy, you are!\n";
	// std::cout << "Write code, you must\n";

	size_t size = vectorSize * sizeof(float);
    float *device_x, *device_y, scale = 2.0f;

    // Initialize host
    std::vector<float> host_x(vectorSize);
    std::vector<float> host_y(vectorSize);
    std::vector<float> host_result(vectorSize);

    // Initialize vector values (vector.data() returns pointer of the first data in the vector)
    vectorInit(host_x.data(), vectorSize);
    vectorInit(host_y.data(), vectorSize);

    // Copy host_y data to host_result vector
    std::memcpy(host_result.data(), host_y.data(), size);

    // Malloc space for x and y in GPU, and use device_x, device_y pointers to point at them
    gpuAssert(hipMalloc((void **)&device_x, size), __FILE__, __LINE__);
    gpuAssert(hipMalloc((void **)&device_y, size), __FILE__, __LINE__);

    // Copy the data from host to GPU
    gpuAssert(hipMemcpy( device_x, host_x.data(), size, hipMemcpyHostToDevice), __FILE__, __LINE__);
    gpuAssert(hipMemcpy( device_y, host_result.data(), size, hipMemcpyHostToDevice), __FILE__, __LINE__);

    // Configure CUDA kernel launch parameters - to have the device to perform actual vector multi-add
    int threadsPerBlock = 256;
    int blocksPerGrid = (vectorSize + threadsPerBlock - 1) / threadsPerBlock;
	// Launch SAXPY kernel
    saxpy_gpu<<<blocksPerGrid, threadsPerBlock>>>(device_x, device_y, scale, vectorSize);
    gpuAssert(hipDeviceSynchronize(), __FILE__, __LINE__);

    // Computation finished, copy the result from GPU to host, verifyVector, free device vectors
    gpuAssert(hipMemcpy(host_result.data(), device_y, size, hipMemcpyDeviceToHost), __FILE__, __LINE__);
    int errorCount = verifyVector(host_x.data(), host_y.data(), host_result.data(), scale, vectorSize);
    std::cout << "Found " << errorCount << " / " << vectorSize << " errors \n";
    gpuAssert(hipFree(device_x), __FILE__, __LINE__);
    gpuAssert(hipFree(device_y), __FILE__, __LINE__);

	return 0;
}

/* 
 Some helpful definitions

 generateThreadCount is the number of threads spawned initially. Each thread is responsible for sampleSize points. 
 *pSums is a pointer to an array that holds the number of 'hit' points for each thread. The length of this array is pSumSize.

 reduceThreadCount is the number of threads used to reduce the partial sums.
 *totals is a pointer to an array that holds reduced values.
 reduceSize is the number of partial sums that each reduceThreadCount reduces.

*/

__global__
void generatePoints (uint64_t * pSums, uint64_t pSumSize, uint64_t sampleSize) {
	//	Insert code here
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= pSumSize) return;
    uint64_t hit_count = 0;
    hiprandState_t rng;
    hiprand_init(clock64(), idx, 0, &rng);
    // int randInt = hiprand_uniform(&rng);
    for(int i = 0; i < sampleSize; i++){
        float x = hiprand_uniform(&rng);
        float y = hiprand_uniform(&rng);
        if((x*x) + (y*y) <= 1.0f){
            ++ hit_count;
        }
    }
    pSums[idx] = hit_count;
}

__global__ 
void reduceCounts (uint64_t * pSums, uint64_t * totals, uint64_t pSumSize, uint64_t reduceSize) {
	//	Insert code here
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	uint64_t reduceThreadCount = pSumSize / reduceSize;
    if(idx >= reduceThreadCount) return;
    uint64_t sum = 0;
    for (uint64_t i = 0; i < reduceSize; i++) {
        sum += pSums[idx * reduceSize + i];
    }
    totals[idx] = sum;
}

int runGpuMCPi (uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {

	//  Check CUDA device presence
	int numDev;
	hipGetDeviceCount(&numDev);
	if (numDev < 1) {
		std::cout << "CUDA device missing!\n";
		return -1;
	}

	auto tStart = std::chrono::high_resolution_clock::now();
		
	float approxPi = estimatePi(generateThreadCount, sampleSize, 
		reduceThreadCount, reduceSize);
	
	std::cout << "Estimated Pi = " << approxPi << "\n";

	auto tEnd= std::chrono::high_resolution_clock::now();

	std::chrono::duration<double> time_span = (tEnd- tStart);
	std::cout << "It took " << time_span.count() << " seconds.";

	return 0;
}

double estimatePi(uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {
	
	double approxPi = 0;

	//      Insert code here
	// std::cout << "Sneaky, you are ...\n";
	// std::cout << "Compute pi, you must!\n";
	size_t generate_size = generateThreadCount * sizeof(uint64_t);
    size_t reduce_size = reduceThreadCount * sizeof(uint64_t);
    std::vector<uint64_t> host_total(reduceThreadCount);
    uint64_t *device_pSums = nullptr;
    uint64_t *device_totals = nullptr;
    gpuAssert(hipMalloc((void **)&device_pSums, generate_size), __FILE__, __LINE__);
    gpuAssert(hipMalloc((void **)&device_totals, reduce_size), __FILE__, __LINE__);

    // Kernel setup for generatePoints (generate random points in GPU)
    int generateThreadsPerBlock = 256;
    int gernerateBlocksPerGrid = (generateThreadCount + generateThreadsPerBlock - 1) / generateThreadsPerBlock;
    generatePoints<<<gernerateBlocksPerGrid, generateThreadsPerBlock>>>(device_pSums, generateThreadCount, sampleSize);
    gpuAssert(hipDeviceSynchronize(), __FILE__, __LINE__);

    // Kernel setup for reduceCounts
    int reduceThreadsPerBlock = 256;
    int reduceBlocksPerGrid = (reduceThreadCount + reduceThreadsPerBlock - 1) / reduceThreadsPerBlock;
    reduceCounts<<<reduceBlocksPerGrid, reduceThreadsPerBlock>>> (device_pSums, device_totals, generateThreadCount, reduceSize);
    gpuAssert(hipDeviceSynchronize(), __FILE__, __LINE__);

    // Computation finished, copy the result from GPU to host
    gpuAssert(hipMemcpy(host_total.data(), device_totals, reduce_size, hipMemcpyDeviceToHost), __FILE__, __LINE__);
    
	uint64_t totalHitCount = 0;
    for (uint64_t i = 0; i < reduceThreadCount; i++) {
        totalHitCount += host_total[i];
    }
    approxPi = (((double)totalHitCount / (double)sampleSize) / (double)generateThreadCount) * 4.0f;

	//Free device vectors
    gpuAssert(hipFree(device_pSums), __FILE__, __LINE__);
    gpuAssert(hipFree(device_totals), __FILE__, __LINE__);
	return approxPi;
}
